#include "hip/hip_runtime.h"
// Add CUDA and C++ Libraries
#include "hip/hip_runtime.h"
#include ""
//for __syncthreads()
#ifndef __cudacc__ 
#define __cudacc__
#endif
#include <stdio.h>
#include <tchar.h>
#include <fstream>
#include <iostream>
#include <limits>
#include <stdlib.h>
#include <math.h>
#include <cmath>
#include <ctime>
using namespace std;
// Define no of rows,cols,frames etc.
int nx = 256;
int ny = 256;
int nt = 11;
double nt_db = nt;
double sumx = 0;
#define PI 3.14159265
void display_matrix_mem_real(double*, int, int);
void H_tempFFT(double*, double*, double*, double*, int, int, int);
__global__ void D_tempFFT(double*, double*, double*, double*, int, int, int, double*, double*, double*, double*, double*, double*, double*, double*, double*, double*);
__global__ void powsum(double*, double*, double*, double*, double*, double*, int, int, int, double);
__global__ void kernel3(double*, double*, double*, double*, double*, double*, int, int, int);
void display_matrix_mem_real(double*, int, int);
void main()
{
	// Allocate memory on host pointers
	double* x_real = (double*)malloc(nx * ny * nt * sizeof(double));
	double* x_imag = (double*)malloc(nx * ny * nt * sizeof(double));

	double* y_real_cpu = (double*)malloc(nx * ny * nt * sizeof(double));
	double* y_imag_cpu = (double*)malloc(nx * ny * nt * sizeof(double));
	double* y_real_gpu = (double*)malloc(nx * ny * nt * sizeof(double));
	double* y_imag_gpu = (double*)malloc(nx * ny * nt * sizeof(double));

	double* shft_fft_b_real_gpu = (double*)malloc(nx * ny * nt * sizeof(double));
	double* shft_fft_b_imag_gpu = (double*)malloc(nx * ny * nt * sizeof(double));
	double* T_M_Lpre_real_gpu= (double*)malloc(nx * ny * nt * sizeof(double));
	double* T_M_Lpre_imag_gpu = (double*)malloc(nx * ny * nt * sizeof(double));

	// Reading text files from directory 
	// Reading Data from the files
	cout << "\t\t opening file to read .......................................................\n";
	ifstream x_read_real("x_real.txt");
	ifstream x_read_imag("x_imag.txt");
	// checking if all the files available in root directory or not
	if (!x_read_real || !x_read_imag)
	{
		cout << "cannot open text files \n";
		(void)getchar();
		return ;
	}
	else
		cout << "all files found:- \t reading data in progress \n";
	//files reading started 
	cout << "\t\t files reading started ......................................................\n";
	double file_in = 0.0;
	cout << "\t\t reading input data .........................................................\n";
	// m real and cmplex 
	for (int i = 0; i < (nx * ny * nt); i++)
	{
		x_read_real >> file_in;
		*(x_real + i) = file_in;
		x_read_imag >> *(x_imag + i);
	}
	double sum_m = 0;
	double e = 2.0;
	for (int i = 0; i < (nx * ny * nt); i++)
	{
		sum_m = sum_m + pow(*(x_real + i), e) + pow(*(x_imag + i), e);
	}
	double norm_m = sqrt(sum_m);
	cout << "\t\t norm_x of input x_ral and x_imag is = " << norm_m << endl;
	// GPU_Implementation_Start
	int start_c = clock();
	H_tempFFT(y_real_cpu, y_imag_cpu, x_real, x_imag, nx, ny, nt);
	int stop_c = clock();
	cout << "\n CPU Time is :" << (stop_c - start_c) << endl;
	// CUDA def
	hipEvent_t start_g, stop_g;
	hipEventCreate(&start_g);
	hipEventCreate(&stop_g);
	// Define Pointers and allocate memory on GPU
	// hipEventRecord(start_g);
	double* d_x_real, * d_x_imag, * d_y_real, * d_y_imag;
	// Pass device pointers 
	double* d_xn_real, * d_xn_imag, * d_n_real, * d_n_imag, * d_k0_real, * d_k0_imag, * d_nk_real, * d_nk_imag, * d_WNnk_real, * d_WNnk_imag;
	hipMalloc((void**)&d_x_real, sizeof(double) * nx * ny * nt);
	hipMalloc((void**)&d_x_imag, sizeof(double) * nx * ny * nt);
	hipMalloc((void**)&d_y_real, sizeof(double) * nx * ny * nt);
	hipMalloc((void**)&d_y_imag, sizeof(double) * nx * ny * nt);
	// Allocate device pointers for 1st kernel processing
	hipMalloc((void**)&d_xn_real, sizeof(double) * nt);
	hipMalloc((void**)&d_xn_imag, sizeof(double) * nt);
	hipMalloc((void**)&d_n_real, sizeof(double) * nt);
	hipMalloc((void**)&d_n_imag, sizeof(double) * nt);
	hipMalloc((void**)&d_k0_real, sizeof(double) * nt);
	hipMalloc((void**)&d_k0_imag, sizeof(double) * nt);
	hipMalloc((void**)&d_nk_real, sizeof(double) * nx * nx * nx);
	hipMalloc((void**)&d_nk_imag, sizeof(double) * nx * nx * nx);
	hipMalloc((void**)&d_WNnk_real, sizeof(double) * nx * nx * nx);
	hipMalloc((void**)&d_WNnk_imag, sizeof(double) * nx * nx * nx);
	// Allocate device pointers for 2nd kernel processing
	double* d_temp1_real, * d_temp1_imag, * d_temp2_real, * d_temp2_imag;
	hipMalloc((void**)&d_temp1_real, sizeof(double) * nx * ny);
	hipMalloc((void**)&d_temp1_imag, sizeof(double) * nx * ny);
	hipMalloc((void**)&d_temp2_real, sizeof(double) * nx * ny);
	hipMalloc((void**)&d_temp2_imag, sizeof(double) * nx * ny);
	// Transfer data from host to device
	hipMemcpy(d_x_real, x_real, sizeof(double) * nx * ny * nt, hipMemcpyHostToDevice);
	hipMemcpy(d_x_imag, x_imag, sizeof(double) * nx * ny * nt, hipMemcpyHostToDevice);
	// first kernel configuration
	dim3 athreads(32, 32); // change
	dim3 ablocks((int)ceil(ny/32), (int)ceil(nx/32));
	// second  kernel configuration
	// dim3 bthreads(nx, ny); // change
	// dim3 bblocks((int)ceil(nt /2), (int)ceil(nt/2));
	hipEventRecord(start_g);
	D_tempFFT<< <ablocks, athreads >> > (d_y_real, d_y_imag, d_x_real, d_x_imag, nx, ny, nt, d_xn_real, d_xn_imag, d_n_real, d_n_imag, d_k0_real, d_k0_imag, d_nk_real, d_nk_imag, d_WNnk_real, d_WNnk_imag);
	hipDeviceSynchronize();
	hipEventRecord(stop_g);
	// powsum << <bblocks, bthreads >> > (d_temp1_real, d_temp1_imag, d_temp2_real, d_temp2_imag, d_y_real, d_y_imag, nx, ny, nt, sumx);
	// hipDeviceSynchronize();
	// int K = nt / 2;
	// kernel3 << <1, K>> > (d_y_real, d_y_imag, d_temp1_real, d_temp1_imag, d_temp2_real, d_temp2_imag, nx, ny, nt);
	// hipDeviceSynchronize();
	// hipEventRecord(stop_g, 0);
   // Copy Result from Device to Host
	hipMemcpy(y_real_gpu, d_y_real, sizeof(double) * nx * ny * nt, hipMemcpyDeviceToHost);
	hipMemcpy(y_imag_gpu, d_y_imag, sizeof(double) * nx * ny * nt, hipMemcpyDeviceToHost);
	// call a CUDA kernel(or launch a kernel);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("CUDA Error: %s\n", hipGetErrorString(err));

		// Possibly: exit(-1) if program cannot continue....
	}
	// Cuda sync
	hipEventRecord(stop_g);
	hipEventSynchronize(stop_g);
	float t = 0;
	hipEventElapsedTime(&t, start_g, stop_g);
	printf("\n GPU time is : %f", t);
	// Free GPU memory
	hipFree(d_x_real);
	hipFree(d_x_imag);
	hipFree(d_y_real);
	hipFree(d_y_imag);
	// Free GPU pointers
	hipFree(d_xn_real);
	hipFree(d_xn_imag);
	hipFree(d_n_real);
	hipFree(d_n_imag);
	hipFree(d_k0_real);
	hipFree(d_k0_imag);
	hipFree(d_nk_real);
	hipFree(d_nk_imag);
	hipFree(d_WNnk_real);
	hipFree(d_WNnk_imag);
	hipFree(d_temp1_real);
	hipFree(d_temp1_imag);
	hipFree(d_temp2_real);
	hipFree(d_temp2_imag);
	// Process further: 
	for (int i = 0; i < nx * ny * nt; i++)
	{
		*(shft_fft_b_real_gpu + i) = *(y_real_gpu + i);
		*(shft_fft_b_imag_gpu + i) = *(y_imag_gpu + i);
	}
	//	display_3d_matrix_mem_complex (shft_fft_b_real, shft_fft_b_imag, nx, ny, nt);
	double sumfft_b = 0;
	for (int i = 0; i < nx * ny * nt; i++)
	{
		sumfft_b = sumfft_b + pow(*(shft_fft_b_real_gpu + i), e) + pow(*(shft_fft_b_imag_gpu + i), e);
	}
	double norm_fft_b = sqrt(sumfft_b);
	cout << "\t\t norm_fft_b at line 455 is = " << norm_fft_b << endl;
	double sqrt_nt;
	sqrt_nt = sqrt(nt_db);

	for (int i = 0; i < nx * ny * nt; i++)
	{
		*(T_M_Lpre_real_gpu + i) = *(shft_fft_b_real_gpu + i) / sqrt_nt;
		*(T_M_Lpre_imag_gpu + i) = *(shft_fft_b_imag_gpu + i) / sqrt_nt;
	}
	// save output data into text files 
	ofstream my_real("out_real.txt");
	ofstream my_imag("out_imag.txt");
	for (int i = 0; i < nx * ny * nt; i++)
	{
		double out_r = 0.0;
		double out_i = 0.0;
		out_r = *(y_real_gpu + i);
		my_real << out_r << endl;
		out_i = *(y_imag_gpu + i);
		my_imag << out_i << endl;
	}
	// close files after saving data
	my_real.close();
	my_imag.close();
	return ;
}
// Temp FFT host function
void H_tempFFT(double* y_real, double* y_imag, double* x_real, double* x_imag, int nx, int ny, int nt)
{
	double WN_real, WN_imag;
	double* xn_real = (double*)malloc(nt * sizeof(double));
	double* xn_imag = (double*)malloc(nt * sizeof(double));
	double* n_real = (double*)malloc(nt * sizeof(double));
	double* n_imag = (double*)malloc(nt * sizeof(double));
	double* k0_real = (double*)malloc(nt * sizeof(double));
	double* k0_imag = (double*)malloc(nt * sizeof(double));
	double* nk_real = (double*)malloc(nx * nx * nx * sizeof(double));
	double* nk_imag = (double*)malloc(nx * nx * nx * sizeof(double));
	double* WNnk_real = (double*)malloc(nx * nx * nx * sizeof(double));
	double* WNnk_imag = (double*)malloc(nx * nx * nx * sizeof(double));
	double* shft_fft_b_real = (double*)malloc(nx * ny * nt * sizeof(double));
	double* shft_fft_b_imag = (double*)malloc(nx * ny * nt * sizeof(double));
	//double* y_real = (double*)malloc(nx * ny * nt * sizeof(double));
	//double* y_imag = (double*)malloc(nx * ny * nt * sizeof(double));
	double* temp1_real = (double*)malloc(nx * ny * nt * sizeof(double));
	double* temp1_imag = (double*)malloc(nx * ny * nt * sizeof(double));
	double* temp2_real = (double*)malloc(nx * ny * sizeof(double));
	double* temp2_imag = (double*)malloc(nx * ny * sizeof(double));
	double e = 2.0;
	for (int c = 0; c < ny; c++)
	{
		for (int r = 0; r < nx; r++)
		{
			for (int i = 0; i < nt; i++)
			{
				*(xn_real + i) = *(x_real + i * nx * ny + r * nx + c);
				*(xn_imag + i) = *(x_imag + i * nx * ny + r * nx + c);
			}

			for (int i = 0; i < nt; i++)
			{
				*(n_real + i) = i;
				*(k0_real + i) = i;
				*(n_imag + i) = i;
				*(k0_imag + i) = i;
			}
			//WN_real = 0.9877;	WN_imag = -0.1564;
			WN_real = cos(2 * PI / nt);	WN_imag = sin(-2 * PI / nt);
			// double e = 2.0;
			double mag = sqrt(pow(WN_real, e) + pow(WN_imag, e));
			double angle = atan(WN_imag / WN_real) * 180 / PI;

			for (int i = 0; i < nt; i++)
			{
				for (int j = 0; j < nt; j++)
				{
					double sum0_real = 0;	double sum0_imag = 0;
					for (int k = 0; k < 1; k++)
					{
						sum0_real = sum0_real + (*(n_real + i + k) * *(k0_real + k + j));
						//sum0_imag = sum0_imag + (*(n_real + i*nt + k) * *(k0_imag + k*nt + j)) + (*(n_imag + i*nt + k) * *(k0_real + k*nt + j));
					}
					*(nk_real + i * nt + j) = sum0_real;
					*(nk_imag + i * nt + j) = sum0_imag;
					*(WNnk_real + i * nt + j) = pow(mag, *(nk_real + i * nt + j)) * cos(angle * sum0_real * PI / 180);		// WNnk (i, j) = WN ^ nk (i, j); Correct this one
					*(WNnk_imag + i * nt + j) = pow(mag, *(nk_real + i * nt + j)) * sin(angle * sum0_real * PI / 180);		// WNnk (i, j) = WN ^ nk (i, j); Correct this one
				}
			}
			//			display_matrix_mem_complex (nk_real, nk_imag, nx*ny, nt);
			//			display_matrix_mem_complex (WNnk_real, WNnk_imag, nx*ny, nt);

			for (int i = 0; i < nt; i++)
			{
				for (int j = 0; j < 1; j++)
				{
					double sum0_real = 0;	double sum0_imag = 0;
					for (int k = 0; k < nt; k++)
					{
						sum0_real = sum0_real + (*(WNnk_real + i * nt + k) * *(xn_real + k + j)) - (*(WNnk_imag + i * nt + k) * *(xn_imag + k + j));
						sum0_imag = sum0_imag + (*(WNnk_real + i * nt + k) * *(xn_imag + k + j)) + (*(WNnk_imag + i * nt + k) * *(xn_real + k + j));
					}
					*(y_real + i * nx * ny + r * nx + c) = sum0_real;
					*(y_imag + i * nx * ny + r * nx + c) = sum0_imag;
				}
			}
		}
	}
}

	//****************************************************//
	/*
	for (int c = 0; c < ny; c++)
	{
		for (int r = 0; r < nx; r++)
		{
			for (int i = 0; i < nt; i++)
			{
				*(xn_real + i) = *(x_real + i * nx * ny + r * nx + c);
				*(xn_imag + i) = *(x_imag + i * nx * ny + r * nx + c);
			}
			//			cout << "xn is: ";
			//			display_matrix_mem_complex (xn_real, xn_imag, 1, nt);

			for (int i = 0; i < nt; i++)
			{
				*(n_real + i) = i;
				*(k0_real + i) = i;
				*(n_imag + i) = i;
				*(k0_imag + i) = i;
			}


			//WN_real = 0.9877;	WN_imag = -0.1564;
			WN_real = cos(2 * PI / nt);	WN_imag = sin(-2 * PI / nt);

			double mag = sqrt(pow(WN_real, e) + pow(WN_imag, e));
			double angle = atan(WN_imag / WN_real) * 180 / PI;

			for (int i = 0; i < nt; i++)
			{
				for (int j = 0; j < nt; j++)
				{
					double sum0_real = 0;	double sum0_imag = 0;
					for (int k = 0; k < 1; k++)
					{
						sum0_real = sum0_real + (*(n_real + i + k) * *(k0_real + k + j));
						//sum0_imag = sum0_imag + (*(n_real + i*nt + k) * *(k0_imag + k*nt + j)) + (*(n_imag + i*nt + k) * *(k0_real + k*nt + j));
					}
					*(nk_real + i * nt + j) = sum0_real;
					*(nk_imag + i * nt + j) = sum0_imag;
					*(WNnk_real + i * nt + j) = pow(mag, *(nk_real + i * nt + j)) * cos(angle * sum0_real * PI / 180);		// WNnk (i, j) = WN ^ nk (i, j); Correct this one
					*(WNnk_imag + i * nt + j) = pow(mag, *(nk_real + i * nt + j)) * sin(angle * sum0_real * PI / 180);		// WNnk (i, j) = WN ^ nk (i, j); Correct this one
				}
			}
			//			display_matrix_mem_complex (nk_real, nk_imag, nx*ny, nt);
			//			display_matrix_mem_complex (WNnk_real, WNnk_imag, nx*ny, nt);

			for (int i = 0; i < nt; i++)
			{
				for (int j = 0; j < 1; j++)
				{
					double sum0_real = 0;	double sum0_imag = 0;
					for (int k = 0; k < nt; k++)
					{
						sum0_real = sum0_real + (*(WNnk_real + i * nt + k) * *(xn_real + k + j)) - (*(WNnk_imag + i * nt + k) * *(xn_imag + k + j));
						sum0_imag = sum0_imag + (*(WNnk_real + i * nt + k) * *(xn_imag + k + j)) + (*(WNnk_imag + i * nt + k) * *(xn_real + k + j));
					}
					*(y_real + i * nx * ny + r * nx + c) = sum0_real;
					*(y_imag + i * nx * ny + r * nx + c) = sum0_imag;
				}
			}
		}
	}
	//	display_3d_matrix_mem_complex (y_real, y_imag, nx, ny, nt);
	sumx = 0;
	for (int i = 0; i < nx * ny * nt; i++)
	{
		sumx = sumx + pow(*(y_real + i), e) + pow(*(y_imag + i), e);
	}

	double norm_x = sqrt(sumx);
	//cout << "\t\t norm_x at line 435 is = " << norm_x << endl;


	int dim = 3, K = nt / 2, len = nt;

	for (int i = 0; i < nx * ny * nt; i++)
	{
		*(temp1_real + i) = 0;
		*(temp1_imag + i) = 0;

		if (i < nx * ny)
		{
			*(temp2_real + i) = 0;
			*(temp2_imag + i) = 0;
		}
	}

	for (int j = 0; j < K; j++)
	{
		int l = 0;
		for (int i = 0; i < nx * ny * nt; i++)
		{
			if (i < nx * ny * (nt - 1))
			{
				*(temp1_real + i) = *(y_real + i);
				*(temp1_imag + i) = *(y_imag + i);
			}
			else
			{
				*(temp2_real + l) = *(y_real + i);
				*(temp2_imag + l) = *(y_imag + i);
				l = l + 1;
			}
		}

		//	display_3d_matrix_mem_complex (temp1_real, temp1_imag, nx, ny, (nt-1));

		for (int i = 0; i < nx * ny * nt; i++)
		{
			if (i < nx * ny)
			{
				*(y_real + i) = *(temp2_real + i);
				*(y_imag + i) = *(temp2_imag + i);
			}
			else
			{
				//				* (X_real + nx*ny + i) = *(temp1_real - nx*ny + i);
				//				* (X_imag + nx*ny + i) = *(temp1_imag - nx*ny + i);
				*(y_real + i) = *(temp1_real - nx * ny + i);
				*(y_imag + i) = *(temp1_imag - nx * ny + i);

			}
		}
	}

	for (int i = 0; i < nx * ny * nt; i++)
	{
		*(shft_fft_b_real + i) = *(y_real + i);
		*(shft_fft_b_imag + i) = *(y_imag + i);
	}

	//	display_3d_matrix_mem_complex (shft_fft_b_real, shft_fft_b_imag, nx, ny, nt);
	double sumfft_b = 0;
	for (int i = 0; i < nx * ny * nt; i++)
	{
		sumfft_b = sumfft_b + pow(*(shft_fft_b_real + i), e) + pow(*(shft_fft_b_imag + i), e);
	}

	double norm_fft_b = sqrt(sumfft_b);
	//cout << "\t\t norm_fft_b at line 455 is = " << norm_fft_b << endl;

	double sqrt_nt;
	sqrt_nt = sqrt(nt_db);

	for (int i = 0; i < nx * ny * nt; i++)
	{
		*(T_M_Lpre_real + i) = *(shft_fft_b_real + i) / sqrt_nt;
		*(T_M_Lpre_imag + i) = *(shft_fft_b_imag + i) / sqrt_nt;
	}
	*/
	//************************************************************//
// Device Function 

__global__ void D_tempFFT(double*y_real, double*y_imag, double*x_real, double*x_imag, int nx, int ny, int nt, double* d_xn_real, double* d_xn_imag, double* d_n_real, double* d_n_imag, double* d_k0_real, double* d_k0_imag, double* d_nk_real, double* d_nk_imag, double* d_WNnk_real, double* d_WNnk_imag)
{
	double WN_real, WN_imag;
	/*double* xn_real = (double*)malloc(nt * sizeof(double));
	double* xn_imag = (double*)malloc(nt * sizeof(double));
	double* n_real = (double*)malloc(nt * sizeof(double));
	double* n_imag = (double*)malloc(nt * sizeof(double));
	double* k0_real = (double*)malloc(nt * sizeof(double));
	double* k0_imag = (double*)malloc(nt * sizeof(double));
	double* nk_real = (double*)malloc(nx * nx * nx * sizeof(double));
	double* nk_imag = (double*)malloc(nx * nx * nx * sizeof(double));
	double* WNnk_real = (double*)malloc(nx * nx * nx * sizeof(double));
	double* WNnk_imag = (double*)malloc(nx * nx * nx * sizeof(double));
	*/
	int c = blockIdx.y * blockDim.y + threadIdx.y;
	int r = blockIdx.x * blockDim.x + threadIdx.x;
	int id = r * nx + c;
	// for (int c = 0; c < ny; c++)
	// {
		//for (int r = 0; r < nx; r++)
		// {
	for (int i = 0; i < nt; i++)
	{
		*(d_xn_real + i) = *(x_real + i * nx * ny + id);
		*(d_xn_imag + i) = *(x_imag + i * nx * ny + id);
	}
	//			cout << "xn is: ";
	//			display_matrix_mem_complex (xn_real, xn_imag, 1, nt);

	for (int i = 0; i < nt; i++)
	{
		*(d_n_real + i) = i;
		*(d_k0_real + i) = i;
		*(d_n_imag + i) = i;
		*(d_k0_imag + i) = i;
	}
	//WN_real = 0.9877;	WN_imag = -0.1564;
	WN_real = cos(2 * PI / nt);	WN_imag = sin(-2 * PI / nt);
	double e = 2.0;
	double mag = sqrt(pow(WN_real, e) + pow(WN_imag, e));
	double angle = atan(WN_imag / WN_real) * 180 / PI;

	for (int i = 0; i < nt; i++)
	{
		for (int j = 0; j < nt; j++)
		{
			double sum0_real = 0;	double sum0_imag = 0;
			for (int k = 0; k < 1; k++)
			{
				sum0_real = sum0_real + (*(d_n_real + i + k) * *(d_k0_real + k + j));
				//sum0_imag = sum0_imag + (*(n_real + i*nt + k) * *(k0_imag + k*nt + j)) + (*(n_imag + i*nt + k) * *(k0_real + k*nt + j));
			}
			*(d_nk_real + i * nt + j) = sum0_real;
			*(d_nk_imag + i * nt + j) = sum0_imag;
			*(d_WNnk_real + i * nt + j) = pow(mag, *(d_nk_real + i * nt + j)) * cos(angle * sum0_real * PI / 180);		// WNnk (i, j) = WN ^ nk (i, j); Correct this one
			*(d_WNnk_imag + i * nt + j) = pow(mag, *(d_nk_real + i * nt + j)) * sin(angle * sum0_real * PI / 180);		// WNnk (i, j) = WN ^ nk (i, j); Correct this one
		}
	}
	//			display_matrix_mem_complex (nk_real, nk_imag, nx*ny, nt);
	//			display_matrix_mem_complex (WNnk_real, WNnk_imag, nx*ny, nt);

	for (int i = 0; i < nt; i++)
	{
		for (int j = 0; j < 1; j++)
		{
			double sum0_real = 0;	double sum0_imag = 0;
			for (int k = 0; k < nt; k++)
			{
				sum0_real = sum0_real + (*(d_WNnk_real + i * nt + k) * *(d_xn_real + k + j)) - (*(d_WNnk_imag + i * nt + k) * *(d_xn_imag + k + j));
				sum0_imag = sum0_imag + (*(d_WNnk_real + i * nt + k) * *(d_xn_imag + k + j)) + (*(d_WNnk_imag + i * nt + k) * *(d_xn_real + k + j));
			}
			*(y_real + i * nx * ny + id) = sum0_real;
			*(y_imag + i * nx * ny + id) = sum0_imag;
		}
	}
	// }
// }
}
//
__global__ void powsum(double* d_temp1_real, double* d_temp1_imag, double* d_temp2_real, double* d_temp2_imag, double* d_y_real, double* d_y_imag, int nx, int ny, int nt, double sumx)
{
	//	double *temp1_real  = (double *) malloc (nx*ny*nt * sizeof(double));
	//	double *temp1_imag  = (double *) malloc (nx*ny*nt * sizeof(double));
	//	double *temp2_real  = (double *) malloc (nx*ny * sizeof(double));
	//	double *temp2_imag  = (double *) malloc (nx*ny * sizeof(double));

	int c = blockIdx.x * blockDim.x + threadIdx.x;
	int r = blockIdx.y * blockDim.y + threadIdx.y;
	int i = c + (r * blockDim.x);
	//	sumx = sumx + pow (*(d_y_real + i), 2) + pow (*(d_y_imag + i), 2);
	sumx = sumx + (*(d_y_real + i) * *(d_y_real + i)) + (*(d_y_imag + i) * *(d_y_imag + i));
	//	for (int i = 0; i<nx*ny*nt; i++)
	//	{
	*(d_temp1_real + i) = 0;
	*(d_temp1_imag + i) = 0;

	if (i < nx * ny)
	{
		*(d_temp2_real + i) = 0;
		*(d_temp2_imag + i) = 0;
	}
}
//
__global__ void kernel3(double* d_y_real, double* d_y_imag, double* d_temp1_real, double* d_temp1_imag, double* d_temp2_real, double* d_temp2_imag, int nx, int ny, int nt)
{
	int j = threadIdx.x;// uncooment when use this kernel 
	int l = 0;
	for (int i = 0; i < nx * ny * nt; i++)
	{
		if (i < nx * ny * (nt - 1))
		{
			*(d_temp1_real + i) = *(d_y_real + i);
			*(d_temp1_imag + i) = *(d_y_imag + i);
		}
		else
		{
			*(d_temp2_real + l) = *(d_y_real + i);
			*(d_temp2_imag + l) = *(d_y_imag + i);
			l = l + 1;
		}
	}
	//	display_3d_matrix_mem_complex (temp1_real, temp1_imag, nx, ny, (nt-1));
	for (int i = 0; i < nx * ny * nt; i++)
	{
		if (i < nx * ny)
		{
			*(d_y_real + i) = *(d_temp2_real + i);
			*(d_y_imag + i) = *(d_temp2_imag + i);
		}
		else
		{
			//				* (X_real + nx*ny + i) = *(temp1_real - nx*ny + i);
			//				* (X_imag + nx*ny + i) = *(temp1_imag - nx*ny + i);
			*(d_y_real + i) = *(d_temp1_real - nx * ny + i);
			*(d_y_imag + i) = *(d_temp1_imag - nx * ny + i);
		}
	}

}

// Display functions 
void display_matrix_mem_real(double* A, int row, int col)
{
	cout << "\n\n";
	for (int i = 0; i < row; i++)
	{
		for (int j = 0; j < col; j++)
		{
			cout << *(A + i * col + j) << "\t";
		}
		cout << "\n";
	}
}
/////////////////////////////////////////////